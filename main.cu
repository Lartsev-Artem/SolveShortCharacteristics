#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>

#include <string>
#include <vector>

#include <random>
#include <iomanip>

#define BS 32
typedef double Type;
using namespace std;

struct Vector3 {
    Type data[3];
    
    __host__ __device__ Type operator [](const int i)  const {
        if (i >= 3) {
            printf("Error Vector3\n");
            return 0;
        }

        return data[i];
    }
    
    Type& operator [](const int i) {
        if (i >= 3) {
            printf("Error Vector3\n");
            return *data;
        }
        return *(data + i);
    }
};

int CheckError(hipError_t cudaStatus, const char* my_text = ""){
    if (cudaStatus != hipSuccess) {
        printf("Error code: %d \n%s \n%s\n", cudaStatus, hipGetErrorString(cudaStatus), my_text);
        return 1;
    }
    return 0;
}

Type NormError(const std::vector<Type>& A, const std::vector<Type>& B) {

    const int N = A.size();
    Type max = -1;
    Type buf;

    for (size_t i = 0; i < N; i++)
    {
        buf = fabs(A[i] - B[i]);
        if (buf > max)
            max = buf;
    }
    return max;
}

size_t ReadSphereDirectionDecartToSpherical(const std::string name_file_sphere_direction, vector<Vector3>& directions_all, vector<Type>& squares, Type& square_surface) {

    std::ifstream ifile;

    ifile.open(name_file_sphere_direction);
    if (!ifile.is_open()) {
        std::cout << "Error read file sphere direction\n";
        return 1;
    }
    int N = 0;
    ifile >> N;
    directions_all.resize(N);
    squares.resize(N);

    for (int i = 0; i < N; i++) {
        ifile >> squares[i];
        ifile >> directions_all[i][0] >> directions_all[i][1] >> directions_all[i][2];
    }
    ifile >> square_surface;
    ifile.close();

    return 0;
}

void GetS(const int N, const int M, const std::vector<Type>& illum_old, vector<Type>& Integ,
    const vector<Vector3>& directions, const vector<Type>& squares, const Type square_surface) {
    //num_cell equals x

    auto Gamma{ [](const Vector3& direction, const Vector3& direction2) {

        Type sum = 0;
        for (size_t i = 0; i < 3; i++)
        {
            sum += direction[i] * direction2[i];
        }

    return (3. * (1 + sum * sum)) / 4;
    } };


    Integ.resize(N * M, 0);

    for (size_t k = 0; k < M; k++)
        for (size_t i = 0; i < N; i++)
        {

            for (int num_direction = 0; num_direction < M; num_direction++)
            {
                Integ[i * M + k] += Gamma(directions[num_direction], directions[k]) *
                    illum_old[i * M + num_direction] * squares[num_direction];
            }

            Integ[i * M + k] / square_surface;

        }
}

__device__ Type Gamma(const Vector3& direction, const Vector3& direction2) {

    Type sum = 0;
    for (size_t i = 0; i < 3; i++)
    {
        sum += direction[i] * direction2[i];
    }

    return (3. * (1 + sum * sum)) / 4;
}

__global__ void d_GetS(const int N, const int M, Type* illum_old, Type* Integ,
    const Type* d_directions, Type* squares, const Type square_surface) {

    const Vector3* directions = (Vector3*)d_directions;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= N) return;
    if (k >= M) return;


    for (int num_direction = 0; num_direction < M; num_direction++)
    {
        Integ[i * M + k] += Gamma(directions[num_direction], directions[k]) *
            illum_old[i * M + num_direction] * squares[num_direction];
    }

    Integ[i * M + k] / square_surface;
}

int main()
{       
    vector<Vector3> directions;
    vector<Type> squares;
    Type square_surface;
    
    ReadSphereDirectionDecartToSpherical("D:\\Desktop\\FilesCourse\\Grids\\Precision\\surface_126_dir.txt", directions, squares, square_surface);
    
    int N = 1e5;
    int M = directions.size();

    std::vector<Type> h_Illum(N * M, 0);

    for (size_t i = 0; i < N * M; i++){
        h_Illum[i] = rand();
    }

    std::vector<Type> Integ(N * M, 0);

    GetS(N, M, h_Illum, Integ, directions, squares, square_surface);

    Type* d_Integ;
    Type* d_directions;
    Type* d_squares;
    Type* d_illum;

    hipEvent_t start, finish;  //��� ������� �������

     // ��������� ������ � ���������� �� device ��������� ������� Y
    {
        //�������� ������� ����� 
        if (CheckError(hipSetDevice(0), "hipSetDevice failed!Do you have a CUDA - capable GPU installed ?")) return 1;


        //======================================buf_device_host==============================================================//

        if (CheckError(hipMalloc(&d_Integ, N * M * sizeof(Type)), "hipMalloc failed!")) return 1;
        if (CheckError(hipMalloc(&d_illum, N * M * sizeof(Type)), "hipMalloc failed!")) return 1;
        if (CheckError(hipMalloc(&d_directions, M * 3 * sizeof(Type)), "hipMalloc failed!")) return 1;
        if (CheckError(hipMalloc(&d_squares, M * sizeof(Type)), "hipMalloc failed!")) return 1;

        //======================================CreateEvent==============================================================//

        hipEventCreate(&start);
        hipEventCreate(&finish);
    }

    // ���������� ������ �� device
    {
        if (CheckError(hipMemcpy(directions.data(), d_directions, M * 3 * sizeof(Type), hipMemcpyHostToDevice), "hipMemcpy failed!")) return 1;
        if (CheckError(hipMemcpy(h_Illum.data(), d_illum, M * N * sizeof(Type), hipMemcpyHostToDevice), "hipMemcpy failed!")) return 1;
        if (CheckError(hipMemcpy(squares.data(), d_squares, M * sizeof(Type), hipMemcpyDeviceToHost), "hipMemcpy failed!")) return 1;
    }

    hipEventRecord(start);
    hipEventSynchronize(start);

   
    dim3 threads(BS, BS);
    dim3 blocks((N + BS - 1) / BS, (M + BS - 1) / BS);

    d_GetS << <blocks, threads >> > (N, M, d_illum, d_Integ, d_directions, d_squares, square_surface);
     
    
    hipEventRecord(finish);
    hipEventSynchronize(finish);

    float time;
    hipEventElapsedTime(&time, start, finish);


    // Check for any errors launching the kernel
    if (CheckError(hipGetLastError(), "calculate_n_body failed!")) return 1;

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
    if (CheckError(hipDeviceSynchronize(), "hipDeviceSynchronize returned error code %d after launching global functions")) return 1;


    std::vector<Type> device_Integ(N * M, 0);
    //����������� �� host 
    {
        if (CheckError(hipMemcpy(device_Integ.data(), d_Integ, N * M * sizeof(Type), hipMemcpyDeviceToHost), "hipMemcpy failed!")) return 1;
    }

    // ����������� ������ � device
    {
        hipEventDestroy(start);
        hipEventDestroy(finish);

        hipFree(d_Integ);
        hipFree(d_squares);
        hipFree(d_directions);
        hipFree(d_illum);

        // hipDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.
        if (CheckError(hipDeviceReset(), "hipDeviceReset failed!")) return 1;
    }

    std::cout << "error host and device:= " << NormError(Integ, device_Integ) << '\n';

    printf("GPU time = %f\n", time / 1000.0f);
    return 0;
}